#include "hip/hip_runtime.h"
#include "src/utils.hpp"

#define min(a, b) a < b ? a : b

__device__ void cross(
    const float* __restrict__ a, 
    const float* __restrict__ b, 
    float* __restrict__ c
) {
    c[0] = a[1] * b[2] - a[2] * b[1];
    c[1] = a[2] * b[0] - a[0] * b[2];
    c[2] = a[0] * b[1] - a[1] * b[0];
}

__device__ void sub(
    const float* __restrict__ a, 
    const float* __restrict__ b, 
    float* __restrict__ c
) {
    c[0] = a[0] - b[0];
    c[1] = a[1] - b[1];
    c[2] = a[2] - b[2];
}

__device__ float dot(const float* __restrict__ a, const float* __restrict__ b) {
    return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}

__device__ float signed_volume(
    const float* __restrict__ a, 
    const float* __restrict__ b, 
    const float* __restrict__ c, 
    const float* __restrict__ d
) {
    float diff_b_a[3];
    float diff_c_a[3];
    float diff_d_a[3];

    float cross_diff_b_a_diff_c_a[3];

    sub(b, a, diff_b_a);
    sub(c, a, diff_c_a);
    sub(d, a, diff_d_a);

    cross(diff_b_a, diff_c_a, cross_diff_b_a_diff_c_a);

    return 1.0 / 6.0 * dot(cross_diff_b_a_diff_c_a, diff_d_a);
}

__device__ bool same_sign(float value_1, float value_2) {
    return (int)(value_1 < 0) == (int)(value_2 < 0);
}

__device__ bool same_sign3(float value_1, float value_2, float value_3) {
    return (value_1 < 0) == (value_2 < 0) && (value_1 < 0) == (value_3 < 0) && (value_2 < 0) == (value_3 < 0);
}

__device__ float signed_area(
    const float * __restrict__ x1, 
    const float * __restrict__ x2, 
    const float * __restrict__ a, 
    const float * __restrict__ w
) {
    float diff_x1_a[3];
    sub(x1, a, diff_x1_a);

    float diff_x2_a[3];
    sub(x2, a, diff_x2_a);

    float cross_diffs[3];
    cross(diff_x1_a, diff_x2_a, cross_diffs);

    return 0.5 * dot(cross_diffs, w);
}



__global__ void watertightness_kernel(
    const float* __restrict__  ray_origins, 
    const float* __restrict__ ray_directions, 
    const float* __restrict__ triangles, 
    float* __restrict__ passed_test,
    int n_rays, 
    int n_triangles
) {
    for (
        int i = blockDim.x * blockIdx.x + threadIdx.x; 
        i < n_rays; 
        i += gridDim.x
    ) {
        int num_intersections = 0;
        for (int triangle_i = 0 ; triangle_i < n_triangles; ++triangle_i) {
            const float *current_triangle = &triangles[triangle_i * 3 * 3];
            const float *p1 = &current_triangle[0];
            const float *p2 = &current_triangle[3];
            const float *p3 = &current_triangle[6];
            
            float a1 = signed_area(p1, p2, &ray_origins[i * 3], &ray_directions[i * 3]);
            float a2 = signed_area(p2, p3, &ray_origins[i * 3], &ray_directions[i * 3]);
            float a3 = signed_area(p3, p1, &ray_origins[i * 3], &ray_directions[i * 3]);

            if (same_sign3(a1, a2, a3)) {
                num_intersections++;
            }
        }
        passed_test[i] = (float)(num_intersections % 2 == 0);
    }
}


void watertightness(
    const float* ray_origins, 
    const float* ray_directions, 
    const float* triangles, 
    float *passed_test,
    int n_rays, 
    int n_triangles,
    hipStream_t stream
) {
    watertightness_kernel<<<65536, 128, 0, stream>>>(
        ray_origins, 
        ray_directions, 
        triangles, 
        passed_test,
        n_rays, 
        n_triangles
    );
    


    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        throw std::runtime_error(
            Formatter() << "CUDA kernel failed : " << std::to_string(err)
        );
    }
}
